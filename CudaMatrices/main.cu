#include "hip/hip_runtime.h"


#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <ctime>
#include <algorithm>
#include <stdio.h>

/// Tile size used by the OptimizedMMKernel
constexpr auto TILE_SIZE = 32;

/// Naive matrix multiplication CUDA Kernel
__global__ void NaiveMMKernel(float *a, float *b, float *c, int size)
{
	int xOut = blockDim.x * blockIdx.x + threadIdx.x;
	int yOut = blockDim.y * blockIdx.y + threadIdx.y;

	float outValue = 0;
	for (int i = 0; i < size; i++)
	{
		// Row of a mulitplied by the column of b
		float prod = a[yOut * size + i] * b[i * size + xOut];
		outValue += prod;
	}

	// Store sum of dot products in C matrix
	c[yOut * size + xOut] = outValue;
}

/// Shared memory matrix multiplication CUDA kernel
__global__ void OptimizedMMKernel(float *a, float *b, float *c, int size)
{
	// Create shared matrices for rows of A and columns of B
	__shared__ float sharedA[TILE_SIZE * TILE_SIZE];
	__shared__ float sharedB[TILE_SIZE * TILE_SIZE];

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int x = blockIdx.x * blockDim.x + tx;
	int y = blockIdx.y * blockDim.y + ty;

	float sum = 0;

	// Divide the matrix up into tiles based on the tile size so each thread
	// Can perform its partial sum of the dot product from the shared matrix
	int tilesPerGrid = size / blockDim.x;
	for (int i = 0; i < tilesPerGrid; i++)
	{
		// Each thread loads element into A and B
		sharedA[ty * TILE_SIZE + tx] = a[(y * size) + (i * TILE_SIZE) + tx];
		sharedB[ty * TILE_SIZE + tx] = b[(i * TILE_SIZE * size) + (ty * size) + x];

		// Wait for all threads to load each section of the shared matrix
		__syncthreads();

		for (int j = 0; j < TILE_SIZE; j++)
		{
			sum += sharedA[ty * TILE_SIZE + j] * sharedB[j * TILE_SIZE + tx];
		}

		// Wait for all threads to compute their partial sum from the shared matrices before loading the next
		__syncthreads();
	}

	// Store the full sum as the result
	c[y * size + x] = sum;
}

/// Prints a matrix out to the stderr stream
void PrintMatrix(float *matrix, int width, int height)
{
	for (int i = 0; i < width * height; i++)
	{
		if (i % width == 0)
		{
			fprintf(stderr, "[ ");
		}

		if (matrix == nullptr)
		{
			fprintf(stderr, "NULL ");
		}
		else
		{
			fprintf(stderr, "%.6f ", matrix[i]);
		}

		if (i % width == width - 1)
		{
			fprintf(stderr, "]\n");
		}
	}
}

/// Checks a cuda call to make sure its OK
void CheckCudaCall(hipError_t callResult, const char *message)
{
	if (callResult != hipSuccess)
	{
		fprintf(stderr, message);
	}
}

/// Calls the naive matrix multiplication kernel
float* NaiveMM(float *a, float *b, int size, int tpb, float *kernelTime, float* totalTime)
{
	const int matrixSizeBytes = size * size * sizeof(float);
	float *result = new float[size * size];
	float *devA, *devB, *devC;

	// Create events
	hipEvent_t kernelStart, kernelStop, totalStart, totalStop;
	CheckCudaCall(hipEventCreate(&kernelStart), "hipEventCreate failed");
	CheckCudaCall(hipEventCreate(&kernelStop), "hipEventCreate failed");
	CheckCudaCall(hipEventCreate(&totalStart), "hipEventCreate failed");
	CheckCudaCall(hipEventCreate(&totalStop), "hipEventCreate failed");

	// Allocate device matrices
 	CheckCudaCall(hipMalloc((void **)&devA, matrixSizeBytes), "hipMalloc failed");
	CheckCudaCall(hipMalloc((void **)&devB, matrixSizeBytes), "hipMalloc failed");
	CheckCudaCall(hipMalloc((void **)&devC, matrixSizeBytes), "hipMalloc failed");

	// Sstart the timer here for the total time
	hipEventRecord(totalStart, 0);

	// Copy over to host
	CheckCudaCall(hipMemcpy(devA, a, matrixSizeBytes, hipMemcpyHostToDevice), "hipMemcpy failed");
	CheckCudaCall(hipMemcpy(devB, b, matrixSizeBytes, hipMemcpyHostToDevice), "hipMemcpy failed");

	// Start the timer here for the kernel time
	hipEventRecord(kernelStart, 0);

	// Call kernel
	dim3 blockSize(size / tpb, size / tpb);
	dim3 threadsPerBlock(tpb, tpb);
	NaiveMMKernel<<<blockSize, threadsPerBlock>>>(devA, devB, devC, size);

	// Stop kernel timer here
	hipEventRecord(kernelStop, 0);
	hipEventSynchronize(kernelStop);

	// Copy result back
	hipMemcpy(result, devC, matrixSizeBytes, hipMemcpyDeviceToHost);

	// Stop total time here
	hipEventRecord(totalStop, 0);
	hipEventSynchronize(totalStop);

	// Calculate elapsed times
	hipEventElapsedTime(kernelTime, kernelStart, kernelStop);
	hipEventElapsedTime(totalTime, totalStart, totalStop);

	// Cleanup device memory
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
	hipFree(kernelStart);
	hipFree(kernelStop);
	hipFree(totalStart);
	hipFree(totalStop);
	
	// Return result
	return result;
}

/// Calls the optimized (shared memory) matrix multiplication kernel
float* OptimizedMM(float *a, float *b, int size, float *kernelTime, float* totalTime)
{
	const int matrixSizeBytes = size * size * sizeof(float);
	float *result = new float[size * size];
	float *devA, *devB, *devC;

	// Create events
	hipEvent_t kernelStart, kernelStop, totalStart, totalStop;
	CheckCudaCall(hipEventCreate(&kernelStart), "hipEventCreate failed");
	CheckCudaCall(hipEventCreate(&kernelStop), "hipEventCreate failed");
	CheckCudaCall(hipEventCreate(&totalStart), "hipEventCreate failed");
	CheckCudaCall(hipEventCreate(&totalStop), "hipEventCreate failed");

	// Allocate device matrices
	CheckCudaCall(hipMalloc((void **)&devA, matrixSizeBytes), "hipMalloc failed");
	CheckCudaCall(hipMalloc((void **)&devB, matrixSizeBytes), "hipMalloc failed");
	CheckCudaCall(hipMalloc((void **)&devC, matrixSizeBytes), "hipMalloc failed");

	// Sstart the timer here for the total time
	hipEventRecord(totalStart, 0);

	// Copy over to host
	CheckCudaCall(hipMemcpy(devA, a, matrixSizeBytes, hipMemcpyHostToDevice), "hipMemcpy failed");
	CheckCudaCall(hipMemcpy(devB, b, matrixSizeBytes, hipMemcpyHostToDevice), "hipMemcpy failed");

	// Start the timer here for the kernel time
	hipEventRecord(kernelStart, 0);

	// Call kernel
	dim3 blockSize(size / TILE_SIZE, size / TILE_SIZE);
	dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
	OptimizedMMKernel <<<blockSize, threadsPerBlock >>>(devA, devB, devC, size);

	// Stop kernel timer here
	hipEventRecord(kernelStop, 0);
	hipEventSynchronize(kernelStop);

	// Copy result back
	hipMemcpy(result, devC, matrixSizeBytes, hipMemcpyDeviceToHost);

	// Stop total time here
	hipEventRecord(totalStop, 0);
	hipEventSynchronize(totalStop);

	// Calculate elapsed times
	hipEventElapsedTime(kernelTime, kernelStart, kernelStop);
	hipEventElapsedTime(totalTime, totalStart, totalStop);

	// Cleanup device memory
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
	hipFree(kernelStart);
	hipFree(kernelStop);
	hipFree(totalStart);
	hipFree(totalStop);

	// Return result
	return result;
}

/// Calls the cublasSgemm function, to multiply 2 matrices
float* CublasMM(hipblasHandle_t &handle, float *a, float *b, int size, float *kernelTime, float* totalTime)
{
	const int matrixSizeBytes = size * size * sizeof(float);
	float *result = new float[size * size];
	float *devA, *devB, *devC;

	// Create events
	hipEvent_t kernelStart, kernelStop, totalStart, totalStop;
	CheckCudaCall(hipEventCreate(&kernelStart), "hipEventCreate failed");
	CheckCudaCall(hipEventCreate(&kernelStop), "hipEventCreate failed");
	CheckCudaCall(hipEventCreate(&totalStart), "hipEventCreate failed");
	CheckCudaCall(hipEventCreate(&totalStop), "hipEventCreate failed");

	// Allocate device matrices
	CheckCudaCall(hipMalloc((void **)&devA, matrixSizeBytes), "hipMalloc failed");
	CheckCudaCall(hipMalloc((void **)&devB, matrixSizeBytes), "hipMalloc failed");
	CheckCudaCall(hipMalloc((void **)&devC, matrixSizeBytes), "hipMalloc failed");

	// Sstart the timer here for the total time
	hipEventRecord(totalStart, 0);

	// Copy over to host
	CheckCudaCall(hipMemcpy(devA, a, matrixSizeBytes, hipMemcpyHostToDevice), "hipMemcpy failed");
	CheckCudaCall(hipMemcpy(devB, b, matrixSizeBytes, hipMemcpyHostToDevice), "hipMemcpy failed");

	// Start the timer here for the kernel time
	hipEventRecord(kernelStart, 0);

	// Initialize cublas params
	const float alp = 1;
	const float bet = 0;
	const float *alpha = &alp;
	const float *beta = &bet;

	// Do the actual multiplication
	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, size, size, size, alpha, devA, size, devB, size, beta, devC, size);

	// Stop kernel timer here
	hipEventRecord(kernelStop, 0);
	hipEventSynchronize(kernelStop);

	// Copy result back
	CheckCudaCall(hipMemcpy(result, devC, matrixSizeBytes, hipMemcpyDeviceToHost), "hipMemcpy failed");

	// Stop total time here
	hipEventRecord(totalStop, 0);
	hipEventSynchronize(totalStop);

	// Calculate elapsed times
	hipEventElapsedTime(kernelTime, kernelStart, kernelStop);
	hipEventElapsedTime(totalTime, totalStart, totalStop);

	// Cleanup device memory
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);
	hipFree(kernelStart);
	hipFree(kernelStop);
	hipFree(totalStart);
	hipFree(totalStop);

	// Return result
	return result;
}

/// Ranomizes a matrix with random floats in the range [0, 5)
void RandomizeMatrix(float *mat, int size)
{
	for (int i = 0; i < size * size; i++) 
	{
		double f = (double)rand() / RAND_MAX;
		mat[i] = (float)(f * 5);
	}
}

/// Sets a matrix's elements as 1..N for testing purposes
void FillMatrixInOrder(float *mat, int size)
{
	for (int i = 0; i < size * size; i++) 
	{
		mat[i] = (float)(i + 1);
	}
}

/// Calculates residual sum between a transposed cuBLAS matrix and a row-major ordered output matrix
float MatrixResidual(float *cublas, float *test, int w, int h)
{
	float dif = 0;
	for (int i = 0; i < w * h; i++)
	{
		// Since cublas matrix is tranposed, swap rows and columns when calculating residual
		int x = i % w;
		int y = i / w;
		int delta = cublas[(x * w) + y] - test[i];
		dif += delta;
	}

	return dif;
}

/// Tests all methods at a specified matrix size
void TestSize(hipblasHandle_t &handle, int size, bool writeOutput)
{
	float *testA = new float[size * size];
	float *testB = new float[size * size];

	FILE *naive_fp; // Naive method file data
	FILE *opt_fp; // Optimized method file data

	if (writeOutput)
	{
		naive_fp = fopen("naive.csv", "a");
		opt_fp = fopen("opt.csv", "a");
	}

	// Randomize matrices
	RandomizeMatrix(testA, size);
	RandomizeMatrix(testB, size);
	printf("Testing size of %i x %i...\n", size, size);

	// Run cublas
	float totalTime0, kernelTime0;
	float* result0 = CublasMM(handle, testA, testB, size, &kernelTime0, &totalTime0);
	printf("Cublas MM | Total (ms) = %.3f | Kernel (ms) = %.3f\n", totalTime0, kernelTime0);

	// Run Naive MM
	const int threadsPerBlock = 32;
	float totalTime1, kernelTime1;
	float* result1 = NaiveMM(testA, testB, size, threadsPerBlock, &kernelTime1, &totalTime1);
	float residual1 = MatrixResidual(result0, result1, size, size);
	printf("Naive MM (%i x %i t/b) | Total (ms) = %.3f | Kernel (ms) = %.3f | Cublas Residual = %.6f\n", threadsPerBlock, threadsPerBlock, totalTime1, kernelTime1, residual1);
	if (writeOutput && naive_fp)
	{
		fprintf(naive_fp, "%i, %.3f, %.3f\n", size, totalTime1, kernelTime1);
	}

	// Run Optimized MM
	float totalTime2, kernelTime2;
	float* result2 = OptimizedMM(testA, testB, size, &kernelTime2, &totalTime2);
	float residual2 = MatrixResidual(result0, result2, size, size);
	printf("Optimized MM (%i x %i t/b) | Total (ms) = %.3f | Kernel (ms) = %.3f | Cublas Residual = %.6f\n", TILE_SIZE, TILE_SIZE, totalTime2, kernelTime2, residual2);
	if (writeOutput && opt_fp)
	{
		fprintf(opt_fp, "%i, %.3f, %.3f\n", size, totalTime2, kernelTime2);
	}

	// Delete input matrices
	delete[] testA;
	delete[] testB;

	// Delete results
	delete[] result0;
	delete[] result1;
	delete[] result2;

	// Close file pointers
	if (writeOutput && naive_fp)
	{
		fclose(naive_fp);
	}

	if (writeOutput && opt_fp)
	{
		fclose(opt_fp);
	}
}

/// Program entry point
int main()
{
	srand(0);

	// Create cublas handle
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	int sizes[8]{ 64, 128, 256, 512, 1024, 2048, 4096, 8192 };
	for (int i = 0; i < 8; i++)
	{
		TestSize(handle, sizes[i], false);
		printf("\n");
	}

	// Destroy the cublas handle
	hipblasDestroy(handle);
}